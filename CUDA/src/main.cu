#include "hip/hip_runtime.h"
/****************************************************************************
*   CUDA Multi Layer Neural Network Implementation.
*
*
*   On linux, compile using nvcc on cuda capable devices
*   Example:
*       $ mkdir build
*       $ nvcc -Iinclude src/main.cu -o build/main
*
*    Or, if you have `cmake` and `make` avaiable
*        - path/to/root/folder$ mkdir build; cd build
*        - path/to/root/folder/build$ cmake ..
*        - path/to/root/folder/build$ make
*    This will create the executable.
*
*
*   Run the executable with:
*   ./cuda [N] [K] [machine output]
*
*   (N = first layer n° neurons)
*   (K = number of layers)
*   (machine output = 1 output machine readable, 0 output human readable. Default: 1)
****************************************************************************/


#include <cstdio>
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <string.h>

#include "hpc.h"


/***************************
 * 
 *  CONSTANT DEFINITION
 * 
****************************/

// Define the "spread index" between layers. Each node is linked to the previous R nodes.
#define R            3

// Define the block dimension for the CUDA kernels
#define BLK_DIM      1024

// Define the BIAS term
#define BIAS         0.2f


__device__ __forceinline__ float sigmoid(float x) {
    /* Sigmoid activation function */
    return 1.0f / (1.0f + expf(-x));
}


inline int layer_size(int N0, int layer) {
    /* Returns the layer-th layer size, given N */
    return N0 - layer * (R - 1);
}


__global__ void forwardpass_shared(
    const float* __restrict__ current_layer,
    float* __restrict__ next_layer,
    const float* __restrict__ W,
    const int N,
    const int out_size
) {
    /* Computes the forward pass using shared memory.
    
    It works by loading the necessary input element into the shared memory before doing
    the forward pass. To do so, it allocate a shared memory region (`s_input`) of dimension
    BLK_DIM + 2 * (R - 1). It must contain at least N elements due to the double
    buffer strategy implemented.
    A more detailed version can be found in the report.pdf in the root folder of the
    project.

    @param current_layer Pointer to the input layer data (activations from the previous layer) in global memory.
    @param next_layer Pointer to the output layer data (activations for the current layer being computed) in global memory.
    @param W Pointer to the weights array in global memory.
    @param N Size (number of elements) of the first input layer (`current_layer`). Used for bounds checking during loading.
    @param out_size Size (number of elements) of the output layer (`next_layer`).
    */

    // shared memory region declaration
    __shared__ float s_input[BLK_DIM + 2 * (R - 1)];

    // halo is being used to load the correct number of elements into the shared memory
    int halo = R - 1;


    int global_index = blockIdx.x * blockDim.x + threadIdx.x;
    int local_index = threadIdx.x;
    int base_index = blockIdx.x * blockDim.x;

    // load the data into the shared memory
    if (global_index < out_size) {
        s_input[local_index] = current_layer[global_index];
    }
    if (local_index < halo) {
        // since out_size < N, but we need additional `halo` elements to corretly do the
        // computation, some thread will also load additional data into the shared memory
        if (global_index + blockDim.x < out_size + R - 1) {
            s_input[local_index + blockDim.x] = current_layer[global_index + blockDim.x];
        }
    }

    __syncthreads();

    // Forward pass
    if (global_index < out_size) {
        float sum = BIAS;
        for (int r = 0; r < R; r++) {
            sum += s_input[local_index + r] * W[base_index + N * r + threadIdx.x];
        }
        next_layer[global_index] = sigmoid(sum);
    }
}

__global__ void forwardpass(
    const float* __restrict__ current_layer,
    float* __restrict__ next_layer,
    const float* __restrict__ W,
    const int N,
    const int out_size
) {
    /* Computes the forward pass.
    
    Similar to the shared memory version, but without the first loading-part.

    @param current_layer Pointer to the input layer data (activations from the previous layer) in global memory.
    @param next_layer Pointer to the output layer data (activations for the current layer being computed) in global memory.
    @param W Pointer to the weights array in global memory.
    @param N Size (number of elements) of the first input layer (`current_layer`). Used for bounds checking during loading.
    @param out_size Size (number of elements) of the output layer (`next_layer`).
    */
    int global_index = blockIdx.x * blockDim.x + threadIdx.x;
    int base_index = blockIdx.x * blockDim.x;
    if (global_index < out_size) {
        float sum = BIAS;
        for (int r = 0; r < R; r++) {
            sum += current_layer[global_index + r] * W[base_index + N * r + threadIdx.x];
        }
        next_layer[global_index] = sigmoid(sum);
    }
}


int main(int argc, char** argv) {
    // Defining the input arguments
    int N = BLK_DIM;
    int K = 2;
    int machine_output = 1;
    if (argc >= 2) N = atoi(argv[1]);
    if (argc >= 3) K = atoi(argv[2]);
    if (argc >= 4) machine_output = atoi(argv[3]);

    // Defining the metrics variables
    double t0, t1, t2, t3, throughput_shared, throughput_no_shared;

    // Allocating the "original" input layer and weight "matrix". Since we will check if
    // both kernels produces the same output, we need to load them using the same randomly
    // sampled data.
    float *original_input = (float*)malloc(N * sizeof(float));
    float *original_W = (float*)malloc(N * R * sizeof(float));

    // Define host pointer to the data
    float *h_input = (float*)malloc(N * sizeof(float));
    float *h_W = (float*)malloc(N * R * sizeof(float));
    float *results_1 = (float*)malloc(N * sizeof(float));
    float *results_2 = (float*)malloc(N * sizeof(float));

    // Filling the original arrays.
    for (int i = 0; i < N; i++) original_input[i] = rand() / (float)RAND_MAX;
    for (int i = 0; i < R * N; i++) original_W[i] = rand() / (float)RAND_MAX;

    // Create and allocate the necessary memory region on the device memory
    float *memory_region;
    hipMalloc(&memory_region, (N + R*N + N) * sizeof(float));

    // Create pointer to the memory region, in order to access it correctly
    float *d_input = memory_region;
    float *d_W = (float*)((char*) memory_region + N * sizeof(float));
    float *d_output =  (float*)((char*) memory_region + (N + R*N) * sizeof(float));

    // Fill the host arrays
    for (int i = 0; i < N; i++) h_input[i] = original_input[i];
    for (int i = 0; i < R * N; i++) h_W[i] = original_W[i];

    // Copy the host array into the device memory.
    hipMemcpy(d_input, h_input, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_W, h_W, N * R * sizeof(float), hipMemcpyHostToDevice);

    t0 = hpc_gettime();
    // Forward pass for the shared kernel
    for (int i=0; i<K; i++) {
        int out_size = layer_size(N, i);
        int blocks = (out_size + BLK_DIM - 1) / BLK_DIM;
        forwardpass_shared<<<blocks, BLK_DIM>>>(d_input, d_output, d_W, N, out_size);
        hipDeviceSynchronize();
        cudaCheckError();

        // Layer swapping
        float* tmp = d_input; d_input = d_output; d_output = tmp;
    }
    t1 = hpc_gettime();

    // Metric computation
    throughput_shared = N / (t1-t0);

    // Copying results into the host pointer.
    hipMemcpy(results_1, d_input, N * sizeof(float), hipMemcpyDeviceToHost);

    // Refilling the host variable with the original data and copying them into the device
    for (int i = 0; i < N; i++) h_input[i] = original_input[i];
    hipMemcpy(d_input, h_input, N * sizeof(float), hipMemcpyHostToDevice);


    t2 = hpc_gettime();
    // Forward pass without shared memory
    for (int i=0; i<K; i++) {
        int out_size = layer_size(N, i);
        int blocks = (out_size + BLK_DIM - 1) / BLK_DIM;
        forwardpass<<<blocks, BLK_DIM>>>(d_input, d_output, d_W, N, out_size);
        hipDeviceSynchronize();
        cudaCheckError();

        // layer swapping
        float* tmp = d_input; d_input = d_output; d_output = tmp;
    }
    t3 = hpc_gettime();

    // Metric computation
    throughput_no_shared = N / (t3-t2);
    hipMemcpy(results_2, d_input, N * sizeof(float), hipMemcpyDeviceToHost);

    // Check if the results are the same (Up to a threshold)
    bool flag = true;
    for (int i = 0; i < layer_size(N, K) and flag; i++) {
        if (abs(results_1[i] - results_2[i]) > 0.001) {
            flag = false;
        }
    }

    char output_control[30];
    if (flag) {
        strcpy(output_control , "Results are equal");
    }
    else {
        strcpy(output_control , "Results differ");
    }

    // Output the results

    if (machine_output)
        //Shared, thrShared, NoShared, thrNoShared, resultsequal
        printf("%f,%f,%f,%f,%i\n", t1-t0, throughput_shared, t3-t2, throughput_no_shared, flag);
    else {
        printf("N = {%i}, K = {%i}, R = {%i}\n", N, K, R);
        printf("Time taken for the no-shared memory kernel: %fs with a throughput of %f elements/second \n", t3 - t2, throughput_no_shared);
        printf("Time taken for the shared memory kernel: %fs with a throughput of %f elements/second \n", t1 - t0, throughput_shared);
        printf("%s between the two kernels!\n",output_control);
    }

    // Free the memory
    hipFree(memory_region);
    free(h_input);
    free(h_W);
    hipDeviceReset();

    return 0;
}
